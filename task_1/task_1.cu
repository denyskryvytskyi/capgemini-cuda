#include "hip/hip_runtime.h"
﻿/**
 * TASK: Addition of two vectors
 * RESULTS: (For vectors with the size = 100'000'000)
 *  - CPU addition: ~110 ms
 *  - GPU addition: ~13 ms
 */

#include "hip/hip_runtime.h"
#include ""

#include <malloc.h>
#include <iostream>
#include <chrono>

constexpr int32_t VEC_SIZE = 100'000'000;
constexpr int32_t ALIGNMENT = 16;
constexpr float VEC_A_OFFSET = 0.2f;
constexpr float VEC_B_OFFSET = 1.3f;
constexpr bool PRINT_VEC = false;

constexpr int CUDA_BLOCK_SIZE = 512;    // amount of threads per block
constexpr int CUDA_NUM_BLOCKS = (VEC_SIZE + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE; // amount of thread blocks

void printVec(float* pVec);
void initData(float* pVecA, float* pVecB);
void add(float* pVecA, float* pVecB, float* pVecRes);
void addWithCuda(float* pDevVecA, float* pDevVecB, float* pDevVecRes, float* pVecRes);

__global__ void addKernel(float* pVecA, float* pVecB, float* pVecRes)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < VEC_SIZE; i += stride)
        pVecRes[i] = pVecA[i] + pVecB[i];
}

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount); // Get the number of CUDA devices

    float* pVecA = static_cast<float*>(_aligned_malloc(VEC_SIZE * sizeof(float), ALIGNMENT));
    if (!pVecA) {
        std::cerr << "Memory allocation failed for vector A." << std::endl;
        return 1;
    }
    
    float* pVecB = static_cast<float*>(_aligned_malloc(VEC_SIZE * sizeof(float), ALIGNMENT));
    if (!pVecB) {
        _aligned_free(pVecA);
        std::cerr << "Memory allocation failed for vector B." << std::endl;
        return 1;
    }
    
    float* pVecRes = static_cast<float*>(_aligned_malloc(VEC_SIZE * sizeof(float), ALIGNMENT));
    if (!pVecRes) {
        _aligned_free(pVecA);
        _aligned_free(pVecB);
        std::cerr << "Memory allocation failed for vector B." << std::endl;
        return 1;
    }

    initData(pVecA, pVecB);

    add(pVecA, pVecB, pVecRes);    // CPU addition

    // Check CUDA GPU device
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipSetDevice failed! CUDA-capable GPU isn't found.\n";
        return 1;
    }

    // CUDA buffers
    float* pDevVecA = nullptr;
    float* pDevVecB = nullptr;
    float* pDevVecRes = nullptr;

    // Allocate GPU buffers for three vectors
    cudaStatus = hipMalloc((void**)&pDevVecA, VEC_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for vector A!\n";
        return 1;
    }

    cudaStatus = hipMalloc((void**)&pDevVecB, VEC_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for vector B!\n";
        hipFree(pDevVecA);
        return 1;
    }

    cudaStatus = hipMalloc((void**)&pDevVecRes, VEC_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for result vector!\n";
        hipFree(pDevVecA);
        hipFree(pDevVecB);
        return 1;
    }

    // Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpy(pDevVecA, pVecA, VEC_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed for vector A!\n";
    }

    cudaStatus = hipMemcpy(pDevVecB, pVecB, VEC_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed for vector B!\n";
    }

    addWithCuda(pDevVecA, pDevVecB, pDevVecRes, pVecRes);

    hipFree(pDevVecA);
    hipFree(pDevVecB);
    hipFree(pDevVecRes);

    _aligned_free(pVecA);
    _aligned_free(pVecB);
    _aligned_free(pVecRes);

    return 0;
}

void printVec(float* pVec)
{
    for (int i = 0; i < VEC_SIZE; ++i) {
        std::cout << pVec[i] << " ";
    }

    std::cout << std::endl;
}

void initData(float* pVecA, float* pVecB)
{
    for (int i = 0; i < VEC_SIZE; ++i) {
        pVecA[i] = static_cast<float>(i) + VEC_A_OFFSET;
        pVecB[i] = static_cast<float>(i) + VEC_B_OFFSET;
    }
}

void add(float* pVecA, float* pVecB, float* pVecRes)
{
    const auto startTimePoint = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < VEC_SIZE; ++i) {
        pVecRes[i] = pVecA[i] + pVecB[i];
    }
    const auto endTimePoint = std::chrono::high_resolution_clock::now();

    if (PRINT_VEC) {
        std::cout << "Result of A + B: ";
        printVec(pVecRes);
    }

    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTimePoint - startTimePoint);
    std::cout << "Execution time: " << duration.count() << " ms.\n";
}

void addWithCuda(float* pDevVecA, float* pDevVecB, float* pDevVecRes, float* pVecRes)
{
    const auto startTimePoint = std::chrono::high_resolution_clock::now();
    addKernel<<<CUDA_NUM_BLOCKS, CUDA_BLOCK_SIZE>>>(pDevVecA, pDevVecB, pDevVecRes);

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cout << "addKernel launch failed:" << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    // hipDeviceSynchronize waits for the kernel to finish
    cudaStatus = hipDeviceSynchronize();
    const auto endTimePoint = std::chrono::high_resolution_clock::now();

    // Any errors encountered during the launch
    if (cudaStatus != hipSuccess) {
        std::cout << "hipDeviceSynchronize returned error code" << cudaStatus << std::endl;
        return;
    }

    // Copy output vector from GPU buffer to host memory
    cudaStatus = hipMemcpy(pVecRes, pDevVecRes, VEC_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed!";
        return;
    }

    if (PRINT_VEC) {
        printVec(pVecRes);
    }

    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTimePoint - startTimePoint);
    std::cout << "Execution time: " << duration.count() << " ms.\n";
}
#include "hip/hip_runtime.h"
﻿/**
 * TASK: Addition of two vectors
 * RESULTS: (For vectors with the size = 100'000'000 with -O3 compilation flag)
 *  - CPU addition: ~110 ms
 *  - GPU addition: ~13 ms
 */

#include "hip/hip_runtime.h"
#include ""

#include <malloc.h>
#include <iostream>
#include <chrono>

constexpr int32_t VEC_SIZE = 100'000'000;
constexpr int32_t ALIGNMENT = 16;
constexpr float VEC_A_OFFSET = 0.2f;
constexpr float VEC_B_OFFSET = 1.3f;
constexpr bool PRINT_VEC = false;

constexpr int32_t CUDA_BLOCK_SIZE = 512;                                                     // amount of threads per threads block
constexpr int32_t CUDA_BLOCKS_PER_GRID = (VEC_SIZE + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE; // amount of thread block

void printVec(float* pVec);
void initData(float* pVecA, float* pVecB);
void cleanup(float* pVecA, float* pVecB, float* pVecRes, float* pDevVecA, float* pDevVecB, float* pDevVecRes);
void add(float* pVecA, float* pVecB, float* pVecRes);
void addWithCuda(float* pDevVecA, float* pDevVecB, float* pDevVecRes, float* pVecRes);

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount); // Get the number of CUDA devices

    if (!deviceCount) {
        std::cout << "CUDA-capable GPU isn't found.\n";
        return 0;
    }

    float* pVecA = static_cast<float*>(_aligned_malloc(VEC_SIZE * sizeof(float), ALIGNMENT));
    if (!pVecA) {
        std::cerr << "Memory allocation failed for vector A." << std::endl;
        return 1;
    }
    
    float* pVecB = static_cast<float*>(_aligned_malloc(VEC_SIZE * sizeof(float), ALIGNMENT));
    if (!pVecB) {
        _aligned_free(pVecA);
        std::cerr << "Memory allocation failed for vector B." << std::endl;
        return 1;
    }
    
    float* pVecRes = static_cast<float*>(_aligned_malloc(VEC_SIZE * sizeof(float), ALIGNMENT));
    if (!pVecRes) {
        _aligned_free(pVecA);
        _aligned_free(pVecB);
        std::cerr << "Memory allocation failed for vector B." << std::endl;
        return 1;
    }

    initData(pVecA, pVecB);

    if (PRINT_VEC) {
        std::cout << "Vector A: ";
        printVec(pVecA);
        
        std::cout << "Vector B: ";
        printVec(pVecB);
    }

    add(pVecA, pVecB, pVecRes);    // CPU addition

    // CUDA buffers
    float* pDevVecA = nullptr;
    float* pDevVecB = nullptr;
    float* pDevVecRes = nullptr;

    // Allocate GPU buffers for three vectors
    hipError_t cudaStatus = hipMalloc(&pDevVecA, VEC_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for vector A!\n";
        cleanup(pVecA, pVecB, pVecRes, pDevVecA, pDevVecB, pDevVecRes);
        return 1;
    }

    cudaStatus = hipMalloc(&pDevVecB, VEC_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for vector B!\n";
        cleanup(pVecA, pVecB, pVecRes, pDevVecA, pDevVecB, pDevVecRes);
        return 1;
    }

    cudaStatus = hipMalloc(&pDevVecRes, VEC_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for result vector!\n";
        cleanup(pVecA, pVecB, pVecRes, pDevVecA, pDevVecB, pDevVecRes);
        return 1;
    }

    // Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpy(pDevVecA, pVecA, VEC_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed for vector A!\n";
        cleanup(pVecA, pVecB, pVecRes, pDevVecA, pDevVecB, pDevVecRes);
        return 1;
    }

    cudaStatus = hipMemcpy(pDevVecB, pVecB, VEC_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed for vector B!\n";
        cleanup(pVecA, pVecB, pVecRes, pDevVecA, pDevVecB, pDevVecRes);
        return 1;
    }

    addWithCuda(pDevVecA, pDevVecB, pDevVecRes, pVecRes);

    cleanup(pVecA, pVecB, pVecRes, pDevVecA, pDevVecB, pDevVecRes);

    return 0;
}

void printVec(float* pVec)
{
    for (int i = 0; i < VEC_SIZE; ++i) {
        std::cout << pVec[i] << " ";
    }

    std::cout << std::endl;
}

void initData(float* pVecA, float* pVecB)
{
    for (int i = 0; i < VEC_SIZE; ++i) {
        pVecA[i] = static_cast<float>(i) + VEC_A_OFFSET;
        pVecB[i] = static_cast<float>(i) + VEC_B_OFFSET;
    }
}

void cleanup(float* pVecA, float* pVecB, float* pVecRes, float* pDevVecA, float* pDevVecB, float* pDevVecRes)
{
    hipFree(pDevVecRes);
    hipFree(pDevVecB);
    hipFree(pDevVecA);

    _aligned_free(pVecRes);
    _aligned_free(pVecB);
    _aligned_free(pVecA);
}

void add(float* pVecA, float* pVecB, float* pVecRes)
{
    std::cout << "===== CPU Addition =====\n";

    const auto startTimePoint = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < VEC_SIZE; ++i) {
        pVecRes[i] = pVecA[i] + pVecB[i];
    }
    const auto endTimePoint = std::chrono::high_resolution_clock::now();

    if (PRINT_VEC) {
        std::cout << "A + B: ";
        printVec(pVecRes);
    }

    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTimePoint - startTimePoint);
    std::cout << "Execution time: " << duration.count() << " ms.\n";
}

__global__ void addKernel(float* pVecA, float* pVecB, float* pVecRes)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (int i = index; i < VEC_SIZE; i += stride)
        pVecRes[i] = pVecA[i] + pVecB[i];
}

void addWithCuda(float* pDevVecA, float* pDevVecB, float* pDevVecRes, float* pVecRes)
{
    std::cout << "===== GPU Addition =====\n";

    const auto startTimePoint = std::chrono::high_resolution_clock::now();
    addKernel<<<CUDA_BLOCKS_PER_GRID, CUDA_BLOCK_SIZE>>>(pDevVecA, pDevVecB, pDevVecRes);

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cout << "addKernel launch failed:" << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    // hipDeviceSynchronize waits for the kernel to finish
    cudaStatus = hipDeviceSynchronize();
    const auto endTimePoint = std::chrono::high_resolution_clock::now();

    // Any errors encountered during the launch
    if (cudaStatus != hipSuccess) {
        std::cout << "hipDeviceSynchronize returned error code" << cudaStatus << std::endl;
        return;
    }

    // Copy output vector from GPU buffer to host memory
    cudaStatus = hipMemcpy(pVecRes, pDevVecRes, VEC_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed!";
        return;
    }

    if (PRINT_VEC) {
        std::cout << "A + B: ";
        printVec(pVecRes);
    }

    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTimePoint - startTimePoint);
    std::cout << "Execution time: " << duration.count() << " ms.\n";
}

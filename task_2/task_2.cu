#include "hip/hip_runtime.h"
/**
 * TASK: Matrix Multiplication Using CUDA
 * RESULTS: (for matrices A(1500; 2000); B(2000; 3000) and result matrix (1500;3000) with -O3 compilation flag)
 *  - CPU multiplication: ~4300 ms
 *  - GPU multiplication: ~540 ms (~8 times faster)
 * TODO:
 *  - tiled matrices
 *  - transpose matrix B for sequential access
 **/

#include <hip/hip_runtime.h>
#include <>

#include <chrono>
#include <iostream>
#include <malloc.h>

constexpr int32_t MAT_DIM_N = 1500; // rows of the matrix A
constexpr int32_t MAT_DIM_M = 2000; // cols of the matrix A and rows of the matrix B
constexpr int32_t MAT_DIM_K = 3000; // cols of the matrix B
constexpr int32_t MAT_A_SIZE = MAT_DIM_N * MAT_DIM_M;
constexpr int32_t MAT_B_SIZE = MAT_DIM_M * MAT_DIM_K;
constexpr int32_t MAT_RES_SIZE = MAT_DIM_N * MAT_DIM_K;
constexpr int32_t ALIGNMENT = 16;
constexpr bool PRINT_MAT = false;
constexpr float MAT_A_OFFSET = 0.5f;
constexpr float MAT_B_OFFSET = 1.3f;

const dim3 THREADS_PER_BLOCK(16, 16);                                       // 16x16 threads per block
const dim3 BLOCKS_PER_GRID((MAT_DIM_K + 15) / 16, (MAT_DIM_N + 15) / 16);   // blocks to cover the matrix

void initData(float* pMatA, float* pMatB, float* pMatRes);
void resetRes(float* pMatRes);
void printMat(float* pMat, int32_t rows, int32_t cols);

void matMul(float* pMatA, float* pMatB, float* pMatRes);
void matMulCuda(float* pDevMatA, float* pDevMatB, float* pDevMatRes, float* pMatRes);

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount); // Get the number of CUDA devices

    if (!deviceCount) {
        std::cout << "CUDA-capable GPU isn't found.\n";
        return 0;
    }

    float* pMatA = static_cast<float*>(_aligned_malloc(MAT_A_SIZE * sizeof(float), ALIGNMENT));
    if (!pMatA) {
        std::cerr << "Failed to allocate memory for matrix A." << std::endl;
        return 1;
    }

    float* pMatB = static_cast<float*>(_aligned_malloc(MAT_B_SIZE * sizeof(float), ALIGNMENT));
    if (!pMatB) {
        _aligned_free(pMatA);
        std::cerr << "Failed to allocate memory for matrix B." << std::endl;
        return 1;
    }

    float* pMatRes = static_cast<float*>(_aligned_malloc(MAT_RES_SIZE * sizeof(float), ALIGNMENT));
    if (!pMatRes) {
        _aligned_free(pMatA);
        _aligned_free(pMatB);
        std::cerr << "Failed to allocate memory for result matrix." << std::endl;
        return 1;
    }

    initData(pMatA, pMatB, pMatRes);

    if (PRINT_MAT) {
        std::cout << "Mat A:\n";
        printMat(pMatA, MAT_DIM_N, MAT_DIM_M);

        std::cout << "Mat B:\n";
        printMat(pMatB, MAT_DIM_M, MAT_DIM_K);
    }

    resetRes(pMatRes);
    matMul(pMatA, pMatB, pMatRes);

    resetRes(pMatRes);

    // CUDA buffers
    float* pDevMatA = nullptr;
    float* pDevMatB = nullptr;
    float* pDevMatRes = nullptr;

    // Allocate GPU buffers for three vectors
    hipError_t cudaStatus = hipMalloc((void**)&pDevMatA, MAT_A_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for vector A!\n";
        return 1;
    }

    cudaStatus = hipMalloc((void**)&pDevMatB, MAT_B_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for vector B!\n";
        hipFree(pDevMatA);
        return 1;
    }

    cudaStatus = hipMalloc((void**)&pDevMatRes, MAT_RES_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for result vector!\n";
        hipFree(pDevMatA);
        hipFree(pDevMatB);
        return 1;
    }

    // Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpy(pDevMatA, pMatA, MAT_A_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed for vector A!\n";
    }

    cudaStatus = hipMemcpy(pDevMatB, pMatB, MAT_B_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed for vector B!\n";
    }

    matMulCuda(pDevMatA, pDevMatB, pDevMatRes, pMatRes);

    hipFree(pDevMatRes);
    hipFree(pDevMatB);
    hipFree(pDevMatA);

    _aligned_free(pMatRes);
    _aligned_free(pMatB);
    _aligned_free(pMatA);

    return 0;
}

void initData(float* pMatA, float* pMatB, float* pMatRes)
{
    for (int i = 0; i < MAT_A_SIZE; ++i) {
        pMatA[i] = static_cast<float>(i) + MAT_A_OFFSET;
    }
    for (int i = 0; i < MAT_B_SIZE; ++i) {
        pMatB[i] = static_cast<float>(i) + MAT_B_OFFSET;
    }
}

void resetRes(float* pMatRes)
{
    memset(pMatRes, 0, MAT_DIM_N * MAT_DIM_K * sizeof(float));
}

void printMat(float* pMat, int32_t rows, int32_t cols)
{
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << pMat[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << std::endl;
}

void matMul(float* pMatA, float* pMatB, float* pMatRes)
{
    std::cout << "===== CPU Matrix Multiplication =====\n";

    // result matrix dim - [N;K]
    const auto startTimePoint = std::chrono::high_resolution_clock::now();
    for (int row = 0; row < MAT_DIM_N; ++row) {
        for (int col = 0; col < MAT_DIM_K; ++col) {
            for (int i = 0; i < MAT_DIM_M; ++i) {
                pMatRes[row * MAT_DIM_K + col] += pMatA[row * MAT_DIM_M + i] * pMatB[i * MAT_DIM_K + col];
            }
        }
    }
    const auto endTimePoint = std::chrono::high_resolution_clock::now();

    if (PRINT_MAT) {
        std::cout << "Result matrix:\n";
        printMat(pMatRes, MAT_DIM_N, MAT_DIM_K);
    }

    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTimePoint - startTimePoint);
    std::cout << "Execution time: " << duration.count() << " ms.\n";
}

__global__ void matMulKernel(float* pMatA, float* pMatB, float* pMatRes, int MAT_DIM_N, int MAT_DIM_M, int MAT_DIM_K)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < MAT_DIM_N && col < MAT_DIM_K) {           // bounding checks
        for (int i = 0; i < MAT_DIM_M; ++i) {
            pMatRes[row * MAT_DIM_K + col] += pMatA[row * MAT_DIM_M + i] * pMatB[i * MAT_DIM_K + col];
        }
    }
}

void matMulCuda(float* pDevMatA, float* pDevMatB, float* pDevMatRes, float* pMatRes)
{
    std::cout << "===== GPU Matrix Multiplication =====\n";

    const auto startTimePoint = std::chrono::high_resolution_clock::now();
    matMulKernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(pDevMatA, pDevMatB, pDevMatRes, MAT_DIM_N, MAT_DIM_M, MAT_DIM_K);

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cout << "matMulKernel launch failed:" << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    cudaStatus = hipDeviceSynchronize();
    const auto endTimePoint = std::chrono::high_resolution_clock::now();

    // Any errors encountered during the launch
    if (cudaStatus != hipSuccess) {
        std::cout << "hipDeviceSynchronize returned error code" << cudaStatus << std::endl;
        return;
    }

    // Copy the result matrix back to the host
    cudaStatus = hipMemcpy(pMatRes, pDevMatRes, MAT_RES_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed!";
        return;
    }

    if (PRINT_MAT) {
        std::cout << "Result matrix:\n";
        printMat(pMatRes, MAT_DIM_N, MAT_DIM_K);
    }

    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTimePoint - startTimePoint);
    std::cout << "Execution time: " << duration.count() << " ms.\n";
}
#include "hip/hip_runtime.h"
/**
 * TASK: Matrix Multiplication Using CUDA
 * NOTE: Implemented and tested on Windows 10 with NVIDIA GTX 1050 (laptop) card
 * RESULTS: (for matrices A(1500; 2000); B(2000; 3000) and result matrix (1500;3000) with -O3 compilation flag)
 *  - CPU multiplication: ~4000 ms
 *  - GPU multiplication (simple kernel matMulKernel): ~200 ms (~20 times faster)
 *  - GPU multiplication (tiled matrix kernel matMulTiledKernel): ~87 ms (~45 times faster)
 *  - GPU data preparation (time to allocate GPU buffers and host to device data copy): ~380 ms
 **/

#include <hip/hip_runtime.h>
#include <>

#include <malloc.h>
#include <iostream>
#include <chrono>

constexpr int32_t MAT_DIM_N = 1500; // rows of the matrix A
constexpr int32_t MAT_DIM_M = 2000; // cols of the matrix A and rows of the matrix B
constexpr int32_t MAT_DIM_K = 3000; // cols of the matrix B
constexpr int32_t MAT_A_SIZE = MAT_DIM_N * MAT_DIM_M;
constexpr int32_t MAT_B_SIZE = MAT_DIM_M * MAT_DIM_K;
constexpr int32_t MAT_RES_SIZE = MAT_DIM_N * MAT_DIM_K;
constexpr int32_t ALIGNMENT = 16;
constexpr float MAT_A_OFFSET = 0.5f;
constexpr float MAT_B_OFFSET = 1.3f;
constexpr bool PRINT_MAT = false;

// CUDA specific
constexpr int32_t TILE_WIDTH = 16;                                          // size of the matrix tile 16x16
constexpr int32_t TILES_AMOUNT = (MAT_DIM_M + TILE_WIDTH - 1) / TILE_WIDTH; // amount of tiles to cover all matrix elements
const dim3 CUDA_BLOCK_SIZE(TILE_WIDTH, TILE_WIDTH);                         // 256 threads per block. Should have the same dimension as matrix tile for efficient processing
const dim3 CUDA_GRID_SIZE((MAT_DIM_K + CUDA_BLOCK_SIZE.x - 1) / CUDA_BLOCK_SIZE.x, (MAT_DIM_N + CUDA_BLOCK_SIZE.y - 1) / CUDA_BLOCK_SIZE.y); // blocks to cover the matrix

// Helpers
void initData(float* pMatA, float* pMatB, float* pMatRes);
void resetRes(float* pMatRes);
void printMat(float* pMat, int32_t rows, int32_t cols);
void cleanup(float* pMatA, float* pMatB, float* pMatRes, float* pDevMatA, float* pDevMatB, float* pDevMatRes);
void matMul(float* pMatA, float* pMatB, float* pMatRes);
void matMulCuda(float* pDevMatA, float* pDevMatB, float* pDevMatRes, float* pMatRes);

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount); // Get the number of CUDA devices

    if (!deviceCount) {
        std::cout << "CUDA-capable GPU isn't found.\n";
        return 0;
    }

    float* pMatA = static_cast<float*>(_aligned_malloc(MAT_A_SIZE * sizeof(float), ALIGNMENT));
    if (!pMatA) {
        std::cerr << "Failed to allocate memory for matrix A." << std::endl;
        return 1;
    }

    float* pMatB = static_cast<float*>(_aligned_malloc(MAT_B_SIZE * sizeof(float), ALIGNMENT));
    if (!pMatB) {
        _aligned_free(pMatA);
        std::cerr << "Failed to allocate memory for matrix B." << std::endl;
        return 1;
    }

    float* pMatRes = static_cast<float*>(_aligned_malloc(MAT_RES_SIZE * sizeof(float), ALIGNMENT));
    if (!pMatRes) {
        _aligned_free(pMatA);
        _aligned_free(pMatB);
        std::cerr << "Failed to allocate memory for result matrix." << std::endl;
        return 1;
    }

    initData(pMatA, pMatB, pMatRes);

    if (PRINT_MAT) {
        std::cout << "Mat A:\n";
        printMat(pMatA, MAT_DIM_N, MAT_DIM_M);

        std::cout << "Mat B:\n";
        printMat(pMatB, MAT_DIM_M, MAT_DIM_K);
    }

    resetRes(pMatRes);
    matMul(pMatA, pMatB, pMatRes);

    std::cout << "===== GPU Matrix Multiplication =====\n";
    resetRes(pMatRes);

    // CUDA buffers
    float* pDevMatA = nullptr;
    float* pDevMatB = nullptr;
    float* pDevMatRes = nullptr;

    const auto startTimePoint = std::chrono::high_resolution_clock::now();

    // Allocate GPU buffers for three vectors
    hipError_t cudaStatus = hipMalloc((void**)&pDevMatA, MAT_A_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for vector A!\n";
        cleanup(pMatA, pMatB, pMatRes, pDevMatA, pDevMatB, pDevMatRes);
        return 1;
    }

    cudaStatus = hipMalloc((void**)&pDevMatB, MAT_B_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for vector B!\n";
        cleanup(pMatA, pMatB, pMatRes, pDevMatA, pDevMatB, pDevMatRes);
        return 1;
    }

    cudaStatus = hipMalloc((void**)&pDevMatRes, MAT_RES_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMalloc failed for result vector!\n";
        cleanup(pMatA, pMatB, pMatRes, pDevMatA, pDevMatB, pDevMatRes);
        return 1;
    }

    // Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpy(pDevMatA, pMatA, MAT_A_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed for vector A!\n";
        cleanup(pMatA, pMatB, pMatRes, pDevMatA, pDevMatB, pDevMatRes);
        return 1;
    }

    cudaStatus = hipMemcpy(pDevMatB, pMatB, MAT_B_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed for vector B!\n";
        cleanup(pMatA, pMatB, pMatRes, pDevMatA, pDevMatB, pDevMatRes);
        return 1;
    }
    const auto endTimePoint = std::chrono::high_resolution_clock::now();

    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTimePoint - startTimePoint);
    std::cout << "GPU data preparation (buffers allocation and host to device data copy) time: " << duration.count() << " ms.\n";

    matMulCuda(pDevMatA, pDevMatB, pDevMatRes, pMatRes);

    cleanup(pMatA, pMatB, pMatRes, pDevMatA, pDevMatB, pDevMatRes);

    return 0;
}

void initData(float* pMatA, float* pMatB, float* pMatRes)
{
    for (int i = 0; i < MAT_A_SIZE; ++i) {
        pMatA[i] = static_cast<float>(i) + MAT_A_OFFSET;
    }
    for (int i = 0; i < MAT_B_SIZE; ++i) {
        pMatB[i] = static_cast<float>(i) + MAT_B_OFFSET;
    }
}

void resetRes(float* pMatRes)
{
    memset(pMatRes, 0, MAT_DIM_N * MAT_DIM_K * sizeof(float));
}

void printMat(float* pMat, int32_t rows, int32_t cols)
{
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << pMat[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << std::endl;
}

void cleanup(float* pMatA, float* pMatB, float* pMatRes, float* pDevMatA, float* pDevMatB, float* pDevMatRes)
{
    hipFree(pDevMatRes);
    hipFree(pDevMatB);
    hipFree(pDevMatA);
    _aligned_free(pMatRes);
    _aligned_free(pMatB);
    _aligned_free(pMatA);
}


void matMul(float* pMatA, float* pMatB, float* pMatRes)
{
    std::cout << "===== CPU Matrix Multiplication =====\n";

    // result matrix dim - [N;K]
    const auto startTimePoint = std::chrono::high_resolution_clock::now();
    for (int row = 0; row < MAT_DIM_N; ++row) {
        for (int col = 0; col < MAT_DIM_K; ++col) {
            for (int i = 0; i < MAT_DIM_M; ++i) {
                pMatRes[row * MAT_DIM_K + col] += pMatA[row * MAT_DIM_M + i] * pMatB[i * MAT_DIM_K + col];
            }
        }
    }
    const auto endTimePoint = std::chrono::high_resolution_clock::now();

    if (PRINT_MAT) {
        std::cout << "Result matrix:\n";
        printMat(pMatRes, MAT_DIM_N, MAT_DIM_K);
    }

    const auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTimePoint - startTimePoint);
    std::cout << "Execution time: " << duration.count() << " ms.\n";
}

__global__ void matMulKernel(float* pMatA, float* pMatB, float* pMatRes, int matDimN, int matDimM, int matDimK)
{
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < matDimN && col < matDimK) {           // bounding checks
        float sum = 0.0f;
        for (int i = 0; i < matDimM; ++i) {
            sum += pMatA[row * matDimM + i] * pMatB[i * matDimK + col];
        }
        pMatRes[row * matDimK + col] = sum;
    }
}

__global__ void matMulTiledKernel(float* pMatA, float* pMatB, float* pMatRes, int matDimN, int matDimM, int matDimK, int tilesAmount)
{
    // Allocate shared memory for sub-matrices A and B tiles for faster access
    // Shared for all threads in one thread block
    __shared__ float sharedMatA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedMatB[TILE_WIDTH][TILE_WIDTH];

    // Calculate element row and column within the output result matrix to be processed by thread
    const int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    const int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // Accumulator for the result element
    float elementValue = 0.0f;

    // One thread block calculates result values for one tile of the result matrix (256 threads in parallel)
    // Loop over tiles of matrices A and B (one tile per one iteration to sum up to element value)
    for (int t = 0; t < tilesAmount; ++t) {
        // Load elements of A and B into shared memory
        const int elementTileAIndex = t * TILE_WIDTH + threadIdx.x; // each thread process one element of the tile from matrix A
        if (row < matDimN && (elementTileAIndex) < matDimM) {
            sharedMatA[threadIdx.y][threadIdx.x] = pMatA[row * matDimM + elementTileAIndex]; 
        } else {
            sharedMatA[threadIdx.y][threadIdx.x] = 0.0f; // Padding for out of bounds threads
        }

        const int elementTileBIndex = t * TILE_WIDTH + threadIdx.y; // each thread process one element of the tile from matrix B
        if (col < matDimK && elementTileBIndex < matDimM) {
            sharedMatB[threadIdx.y][threadIdx.x] = pMatB[elementTileBIndex * matDimK + col];
        } else {
            sharedMatB[threadIdx.y][threadIdx.x] = 0.0f; // Padding for out of bounds threads
        }

        // Synchronize to make sure all data is loaded before computation
        __syncthreads();

        // Perform computation on the tile
        for (int i = 0; i < TILE_WIDTH; ++i) {
            elementValue += sharedMatA[threadIdx.y][i] * sharedMatB[i][threadIdx.x];
        }

        // Synchronize to make sure all threads are done before loading the next tile
        __syncthreads();
    }

    // Write the result back to the output matrix C
    if (row < matDimN && col < matDimK) {
        pMatRes[row * matDimK + col] = elementValue;
    }
}

void matMulCuda(float* pDevMatA, float* pDevMatB, float* pDevMatRes, float* pMatRes)
{
    hipEvent_t startKernelEvent, stopKernelEvent; // events to measure kernel execution time
    hipEventCreate(&startKernelEvent);
    hipEventCreate(&stopKernelEvent);

    hipEventRecord(startKernelEvent, 0);
    matMulTiledKernel<<<CUDA_GRID_SIZE, CUDA_BLOCK_SIZE>>>(pDevMatA, pDevMatB, pDevMatRes, MAT_DIM_N, MAT_DIM_M, MAT_DIM_K, TILES_AMOUNT);

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cout << "matMulKernel launch failed:" << hipGetErrorString(cudaStatus) << std::endl;
        return;
    }

    hipEventRecord(stopKernelEvent, 0);
    hipEventSynchronize(stopKernelEvent);

    float kernelTimeMs = 0.0f;
    hipEventElapsedTime(&kernelTimeMs, startKernelEvent, stopKernelEvent);
    hipEventDestroy(startKernelEvent);
    hipEventDestroy(stopKernelEvent);

    cudaStatus = hipDeviceSynchronize();

    // Any errors encountered during the launch
    if (cudaStatus != hipSuccess) {
        std::cout << "hipDeviceSynchronize returned error code" << cudaStatus << std::endl;
        return;
    }

    // Copy the result matrix back to the host
    cudaStatus = hipMemcpy(pMatRes, pDevMatRes, MAT_RES_SIZE * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cout << "hipMemcpy failed!";
        return;
    }

    if (PRINT_MAT) {
        std::cout << "Result matrix:\n";
        printMat(pMatRes, MAT_DIM_N, MAT_DIM_K);
    }

    std::cout << "Execution time (kernel): " << kernelTimeMs << " ms.\n";
}